
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <chrono>

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

constexpr int tile_width = 16;
constexpr int samples_to_check = 10000;

__global__ void tiled_matrix_multiply(float *d_A, float *d_B, float *d_C, int A_rows, int A_cols, int B_rows, int B_cols)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  int num_tiles = (A_cols + tile_width - 1) / tile_width;
  __shared__ float A_tile[tile_width][tile_width];
  __shared__ float B_tile[tile_width][tile_width];

  float sum = 0;
  for (int i = 0; i < num_tiles; i++)
  {
    int A_read_row = row;
    int A_read_col = tile_width * i + threadIdx.x;

    if (A_read_row < A_rows && A_read_col < A_cols)
    {
      A_tile[threadIdx.y][threadIdx.x] = d_A[A_read_row * A_cols + A_read_col];
    }
    else
    {
      A_tile[threadIdx.y][threadIdx.x] = 0;
    }

    int B_read_row = tile_width * i + threadIdx.y;
    int B_read_col = col;

    if (B_read_row < B_rows && B_read_col < B_cols)
    {
      B_tile[threadIdx.y][threadIdx.x] = d_B[B_read_row * B_cols + B_read_col];
    }
    else
    {
      B_tile[threadIdx.y][threadIdx.x] = 0;
    }

    __syncthreads();

    for (int k = 0; k < tile_width; k++)
    {
      sum += A_tile[threadIdx.y][k] * B_tile[k][threadIdx.x];
    }

    __syncthreads();
  }

  if (row < A_rows && col < B_cols)
  {
    d_C[row * B_cols + col] = sum;
  }
}

int verify_matrix_multiplication(float *h_A, float *h_B, float *h_C, int A_rows, int A_cols, int B_rows, int B_cols)
{
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);

  std::uniform_int_distribution<> row_dist(0, A_rows - 1);
  std::uniform_int_distribution<> col_dist(0, B_cols - 1);

  for (int s = 0; s < samples_to_check; s++)
  {
    int i = row_dist(generator);
    int j = col_dist(generator);

    float target_value = h_C[i * B_cols + j];
    float expected_value = 0.0f;

    for (int k = 0; k < A_cols; k++)
    {
      expected_value += h_A[i * A_cols + k] * h_B[k * B_cols + j];
    }

    if (fabs(target_value - expected_value) > 1e-5)
    {
      printf("Mismatch (%d, %d): expected %f, got %f\n", i, j, expected_value, target_value);
      return 1;
    }
  }

  return 0;
}

int main()
{
  // Create random number generator and random distribution
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);
  std::uniform_real_distribution<float> distribution(-0.5f, 0.5f);

  // Define matrix A with values from a random distribution
  int A_rows = 3000;
  int A_cols = 2000;
  size_t A_memsize = A_rows * A_cols * sizeof(float);
  float *h_A = (float *)malloc(A_memsize);

  for (int i = 0; i < A_rows; i++)
  {
    for (int j = 0; j < A_cols; j++)
    {
      h_A[i * A_cols + j] = distribution(generator);
    }
  }

  // Define matrix B values from a random distribution
  int B_rows = 2000;
  int B_cols = 3000;
  size_t B_memsize = B_rows * B_cols * sizeof(float);
  float *h_B = (float *)malloc(B_memsize);

  for (int i = 0; i < B_rows; i++)
  {
    for (int j = 0; j < B_cols; j++)
    {
      h_B[i * B_cols + j] = distribution(generator);
    }
  }

  // Allocate memory for matrix C which will be the product of A and B
  int C_rows = A_rows;
  int C_cols = B_cols;
  size_t C_memsize = C_rows * C_cols * sizeof(float);
  float *h_C = (float *)malloc(C_memsize);

  // Prepare device variables for the matrices
  float *d_A, *d_B, *d_C;
  CUDA_CHECK(hipMalloc((void **)&d_A, A_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_B, B_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_C, C_memsize));

  // Move data from host to device
  CUDA_CHECK(hipMemcpy(d_A, h_A, A_memsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, h_B, B_memsize, hipMemcpyHostToDevice));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Compute the number of blocks needed
  dim3 threads_per_block(tile_width, tile_width);
  int blocks_x = (C_cols + threads_per_block.x - 1) / threads_per_block.x;
  int blocks_y = (C_rows + threads_per_block.y - 1) / threads_per_block.y;
  dim3 num_blocks(blocks_x, blocks_y);

  // Execute the kernel
  tiled_matrix_multiply<<<num_blocks, threads_per_block>>>(d_A, d_B, d_C, A_rows, A_cols, B_rows, B_cols);
  CUDA_CHECK(hipGetLastError());

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel execution time: %f ms\n", milliseconds);

  // Move data from device to host
  CUDA_CHECK(hipMemcpy(h_C, d_C, C_memsize, hipMemcpyDeviceToHost));

  // Check values
  if (verify_matrix_multiplication(h_A, h_B, h_C, A_rows, A_cols, B_rows, B_cols) != 0)
  {
    return 1;
  }
  printf("All (sampled) values match\n");

  // Free memory and destroy events
  free(h_A);
  free(h_B);
  free(h_C);

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}