#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb/stb_image.h"
#include "stb/stb_image_write.h"

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

constexpr int blur_radius = 135;
constexpr dim3 threads_per_block(16, 16, 1);

__global__ void blur_kernel(uchar3 *d_input_image, uchar3 *d_output_image, int width, int height)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    float3 sum = {0.0f, 0.0f, 0.0f};
    int count = 0;

    for (int dx = -blur_radius; dx <= blur_radius; dx++)
    {
      for (int dy = -blur_radius; dy <= blur_radius; dy++)
      {

        int neighbor_col = col + dx;
        int neighbor_row = row + dy;

        if (-1 < neighbor_col && neighbor_col < width && -1 < neighbor_row && neighbor_row < height)
        {
          count += 1;

          int neighbor_idx = neighbor_row * width + neighbor_col;
          uchar3 neighbor_pixel = d_input_image[neighbor_idx];

          sum.x += neighbor_pixel.x;
          sum.y += neighbor_pixel.y;
          sum.z += neighbor_pixel.z;
        }
      }
    }

    int output_idx = row * width + col;
    d_output_image[output_idx] = make_uchar3(sum.x / count, sum.y / count, sum.z / count);
  }
}

int main(int argc, char *argv[])
{
  if (argc != 3)
  {
    fprintf(stderr, "Usage: %s <input_file> <output_file>\n", argv[0]);
    return 1;
  }

  const char *input_filename = argv[1];
  const char *output_filename = argv[2];

  printf("Will read from %s and write to %s\n", input_filename, output_filename);

  // Load input image from file
  int width, height, channels;
  unsigned char *h_input_image_stb = stbi_load(input_filename, &width, &height, &channels, 3);
  if (h_input_image_stb == NULL)
  {
    fprintf(stderr, "Error loading image: %s\n", stbi_failure_reason());
    return 1;
  }

  printf("Loaded image %s (%d x %d)\n", input_filename, width, height);

  size_t mem_size = (size_t)width * height * 3 * sizeof(unsigned char);

  // Define host input and output image vectors
  std::vector<unsigned char> h_input_image(h_input_image_stb, h_input_image_stb + mem_size);
  std::vector<unsigned char> h_output_image(mem_size);

  stbi_image_free(h_input_image_stb);

  // Prepare device variables for input and output images
  unsigned char *d_input_image;
  unsigned char *d_output_image;
  CUDA_CHECK(hipMalloc((void **)&d_input_image, mem_size));
  CUDA_CHECK(hipMalloc((void **)&d_output_image, mem_size));

  // Move input image data from host to device
  CUDA_CHECK(hipMemcpy(d_input_image, h_input_image.data(), mem_size, hipMemcpyHostToDevice));

  // Perform image blur on GPU
  int blocks_x = (width + threads_per_block.x - 1) / threads_per_block.x;
  int blocks_y = (height + threads_per_block.y - 1) / threads_per_block.y;
  dim3 num_blocks(blocks_x, blocks_y);
  blur_kernel<<<num_blocks, threads_per_block>>>((uchar3 *)d_input_image, (uchar3 *)d_output_image, width, height);
  CUDA_CHECK(hipGetLastError());

  // Move output image data from device to host
  CUDA_CHECK(hipMemcpy(h_output_image.data(), d_output_image, mem_size, hipMemcpyDeviceToHost));

  // Write output image to file
  int success = stbi_write_png(output_filename, width, height, 3, h_output_image.data(), width * 3);
  if (!success)
  {
    fprintf(stderr, "Error writing image %s\n", output_filename);
  }
  else
  {
    printf("Successfully wrote image to %s\n", output_filename);
  }

  // Free device memory
  CUDA_CHECK(hipFree(d_input_image));
  CUDA_CHECK(hipFree(d_output_image));

  return 0;
}