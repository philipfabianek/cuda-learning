
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <chrono>

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

#define MATRIX_ROWS 30000
#define MATRIX_COLS 20000
#define RANDOM_INITIALIZATION false
#define FILTER_RADIUS_X 2
#define FILTER_RADIUS_Y 2
#define FILTER_ROWS (2 * FILTER_RADIUS_Y + 1)
#define FILTER_COLS (2 * FILTER_RADIUS_X + 1)
#define INPUT_TILE_SIZE 16
#define OUTPUT_TILE_SIZE_X (INPUT_TILE_SIZE - 2 * FILTER_RADIUS_X)
#define OUTPUT_TILE_SIZE_Y (INPUT_TILE_SIZE - 2 * FILTER_RADIUS_Y)
#define SAMPLES_TO_CHECK 10000

__constant__ float d_F[2 * FILTER_RADIUS_Y + 1][2 * FILTER_RADIUS_X + 1];

/*
 * Performs a 2D convolution using a tiled memory approach.
 * - Each thread block is responsible for one output tile.
 * - To compute the output tile, the block loads a larger corresponding input tile into shared memory.
 *   The halo region accounts for the filter radius.
 * - The filter coefficients are stored in constant memory for faster access.
 */
__global__ void tiled_convolution(float *d_A, float *d_B, int matrix_rows, int matrix_cols)
{
  signed int col = blockIdx.x * OUTPUT_TILE_SIZE_X + threadIdx.x - FILTER_RADIUS_X;
  signed int row = blockIdx.y * OUTPUT_TILE_SIZE_Y + threadIdx.y - FILTER_RADIUS_Y;

  __shared__ float input_tile[INPUT_TILE_SIZE][INPUT_TILE_SIZE];

  // Load input tile into shared memory using all threads in the block
  if (row >= 0 && row < matrix_rows && col >= 0 && col < matrix_cols)
  {
    input_tile[threadIdx.y][threadIdx.x] = d_A[row * matrix_cols + col];
  }
  else
  {
    input_tile[threadIdx.y][threadIdx.x] = 0.0f;
  }

  __syncthreads();

  // Compute indices of the thread inside the output tile
  signed int output_tile_x = threadIdx.x - FILTER_RADIUS_X;
  signed int output_tile_y = threadIdx.y - FILTER_RADIUS_Y;

  // Check whether the thread should be enabled for the current output tile
  // (threads on the edges are disabled because input tiles are bigger than output tiles)
  if (output_tile_x >= 0 && output_tile_x < OUTPUT_TILE_SIZE_X && output_tile_y >= 0 && output_tile_y < OUTPUT_TILE_SIZE_Y)
  {
    // Check that thread indice s are within bounds,
    // intuitively it is better to have this check nested inside the other one
    // and the profiler agrees (there are ~ 5% less elapsed cycles in the Nsight Compute)
    if (row >= 0 && row < matrix_rows && col >= 0 && col < matrix_cols)
    {
      float sum = 0.0f;

      for (int i = -FILTER_RADIUS_Y; i < FILTER_RADIUS_Y + 1; i++)
      {
        for (int j = -FILTER_RADIUS_X; j < FILTER_RADIUS_X + 1; j++)
        {
          sum += d_F[FILTER_RADIUS_Y + i][FILTER_RADIUS_X + j] * input_tile[threadIdx.y + i][threadIdx.x + j];
        }
      }

      d_B[row * matrix_cols + col] = sum;
    }
  }
}

int verify_convolution(float *h_A, float *h_B, float *h_F, int matrix_rows, int matrix_cols)
{
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);

  std::uniform_int_distribution<> row_dist(0, matrix_rows - 1);
  std::uniform_int_distribution<> col_dist(0, matrix_cols - 1);

  for (int s = 0; s < SAMPLES_TO_CHECK; s++)
  {
    int i = row_dist(generator);
    int j = col_dist(generator);

    float target_value = h_B[i * matrix_cols + j];
    float expected_value = 0.0f;

    for (int k = -FILTER_RADIUS_Y; k < FILTER_RADIUS_Y + 1; k++)
    {
      for (int l = -FILTER_RADIUS_X; l < FILTER_RADIUS_X + 1; l++)
      {
        if (i + k >= 0 && i + k < matrix_rows && j + l >= 0 && j + l < matrix_cols)
        {
          expected_value += h_A[(i + k) * matrix_cols + (j + l)] * h_F[(FILTER_RADIUS_Y + k) * FILTER_COLS + (FILTER_RADIUS_X + l)];
        }
      }
    }

    if (fabs(target_value - expected_value) > 1e-5)
    {
      printf("Mismatch (%d, %d): expected %f, got %f\n", i, j, expected_value, target_value);
      return 1;
    }
  }

  return 0;
}

int main()
{
  // Create random number generator and random distribution
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);
  std::uniform_real_distribution<float> distribution(-0.5f, 0.5f);

  // Define matrix A with values from a random distribution
  int matrix_rows = MATRIX_ROWS;
  int matrix_cols = MATRIX_COLS;
  size_t A_memsize = matrix_rows * matrix_cols * sizeof(float);
  float *h_A = (float *)malloc(A_memsize);

  for (int i = 0; i < matrix_rows; i++)
  {
    for (int j = 0; j < matrix_cols; j++)
    {
      if (RANDOM_INITIALIZATION)
      {
        h_A[i * matrix_cols + j] = distribution(generator);
      }
      else
      {
        h_A[i * matrix_cols + j] = 1.0f;
      }
    }
  }

  // Define filter F with values from a random distribution
  size_t F_memsize = FILTER_ROWS * FILTER_COLS * sizeof(float);
  float *h_F = (float *)malloc(F_memsize);

  for (int i = 0; i < FILTER_ROWS; i++)
  {
    for (int j = 0; j < FILTER_COLS; j++)
    {
      if (RANDOM_INITIALIZATION)
        h_F[i * FILTER_COLS + j] = distribution(generator);
      else
      {
        h_F[i * FILTER_COLS + j] = 1.0f;
      }
    }
  }

  // Allocate memory for matrix B which will be the result of the convolution
  // (formally cross-correlation) of the matrix A with the filter F
  // (it will have same size as A, zeros will be used as padding)
  size_t B_memsize = A_memsize;
  float *h_B = (float *)malloc(B_memsize);

  // Prepare device variables for the matrices
  float *d_A, *d_B;
  CUDA_CHECK(hipMalloc((void **)&d_A, A_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_B, B_memsize));

  // Move data from host to device
  CUDA_CHECK(hipMemcpy(d_A, h_A, A_memsize, hipMemcpyHostToDevice));

  // Copy filter F to constant memory
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_F), h_F, F_memsize));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Perform convolution on GPU
  dim3 threads_per_block(INPUT_TILE_SIZE, INPUT_TILE_SIZE);
  int blocks_x = (matrix_cols + OUTPUT_TILE_SIZE_X - 1) / OUTPUT_TILE_SIZE_X;
  int blocks_y = (matrix_rows + OUTPUT_TILE_SIZE_Y - 1) / OUTPUT_TILE_SIZE_Y;
  dim3 num_blocks(blocks_x, blocks_y);
  tiled_convolution<<<num_blocks, threads_per_block>>>(d_A, d_B, matrix_rows, matrix_cols);
  CUDA_CHECK(hipGetLastError());

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel execution time: %f ms\n", milliseconds);

  // Move data from device to host
  CUDA_CHECK(hipMemcpy(h_B, d_B, B_memsize, hipMemcpyDeviceToHost));

  // Check values
  if (verify_convolution(h_A, h_B, h_F, matrix_rows, matrix_cols) != 0)
  {
    return 1;
  }
  printf("All (sampled) values match\n");

  // Free memory and destroy events
  free(h_A);
  free(h_B);
  free(h_F);

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}