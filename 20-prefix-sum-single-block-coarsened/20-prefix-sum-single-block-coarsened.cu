
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <random>
#include <chrono>

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

// This program works with only one block, so only up to 1024 threads,
// it is also limited by the shared memory so N cannot be too large.
constexpr int threads_per_block = 256;
constexpr int blocks_per_grid = 1;
constexpr int coarse_factor = 4;
constexpr int N = coarse_factor * threads_per_block;

/*
 * Performs prefix sum using shared memory, Kogge-Stone algorithm and thread coarsening.
 * - In the first part, data is loaded into shared memory and each thread
 *   performs prefix sum on its chunk, which is a segment consisting of <coarse_factor> elements.
 * - In the second part, the Kogge-Stone prefix sum algorithm is applied
 *   to the last elements of each chunk.
 * - In the third part, each chunk is updated by adding the last element of the previous chunk
 *   and the results are written back to the global memory.
 */
__global__ void prefix_sum_kernel(unsigned int *d_input_array, unsigned int *d_sum_result)
{
  unsigned int tx = threadIdx.x;
  unsigned int last_idx = (tx + 1) * coarse_factor - 1;
  int prev_last_idx = last_idx - coarse_factor;

  __shared__ unsigned int temp_result[N];

  // Load data into shared memory in a coalesced manner
  for (int i = 0; i < coarse_factor; i++)
  {
    int idx = tx + i * threads_per_block;
    temp_result[idx] = d_input_array[idx];
  }

  __syncthreads();

  // Perform prefix sum on each chunk
  for (int i = 1; i < coarse_factor; i++)
  {
    int idx = tx * coarse_factor + i;
    temp_result[idx] += temp_result[idx - 1];
  }

  // Perform the Kogge-Stone prefix sum algorithm
  // on the last elements of each chunk
  unsigned int temp;
  for (int stride = coarse_factor; stride < N; stride *= 2)
  {
    __syncthreads();

    if (last_idx >= stride)
    {
      temp = temp_result[last_idx - stride] + temp_result[last_idx];
    }

    __syncthreads();

    if (last_idx >= stride)
    {
      temp_result[last_idx] = temp;
    }
  }

  __syncthreads();

  // For each chunk, add the last element of the previous chunk
  if (prev_last_idx >= 0)
  {
    for (int i = 0; i < coarse_factor - 1; i++)
    {
      int idx = tx * coarse_factor + i;
      temp_result[idx] += temp_result[prev_last_idx];
    }
  }

  __syncthreads();

  // Write the results back to the global memory
  for (int i = 0; i < coarse_factor; i++)
  {
    int idx = tx + i * threads_per_block;
    d_sum_result[idx] = temp_result[idx];
  }
}

int verify_prefix_sum(const unsigned int *h_input_array, const unsigned int *h_gpu_sum)
{
  std::vector<unsigned int> target_sum(N);

  clock_t start_time = clock();

  target_sum[0] = h_input_array[0];

  for (int i = 1; i < N; ++i)
  {
    target_sum[i] = target_sum[i - 1] + h_input_array[i];
  }

  clock_t end_time = clock();
  double elapsed_time = double(end_time - start_time) / CLOCKS_PER_SEC;
  printf("CPU verification time: %f seconds\n", elapsed_time);

  for (int i = 0; i < N; ++i)
  {
    if (target_sum[i] != h_gpu_sum[i])
    {
      printf("Mismatch at index %d: expected %d, got %d\n", i, target_sum[i], h_gpu_sum[i]);
      return 1;
    }
  }

  return 0;
}

int main()
{
  // Create random number generator and random distribution
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);
  std::uniform_int_distribution<unsigned int> distribution(0, 4);

  // Define input array with values from the random distribution
  size_t input_memsize = N * sizeof(unsigned int);
  std::vector<unsigned int> h_input_array(N);

  for (int i = 0; i < N; ++i)
  {
    h_input_array[i] = distribution(generator);
  }

  // Allocate memory for the host prefix sum result
  size_t sum_memsize = N * sizeof(unsigned int);
  std::vector<unsigned int> h_sum_result(N);

  // Prepare device variables
  unsigned int *d_input_array;
  unsigned int *d_sum_result;
  CUDA_CHECK(hipMalloc((void **)&d_input_array, input_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_sum_result, sum_memsize));

  // Move data from host to device
  CUDA_CHECK(hipMemcpy(d_input_array, h_input_array.data(), input_memsize, hipMemcpyHostToDevice));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Compute prefix sum on GPU
  prefix_sum_kernel<<<blocks_per_grid, threads_per_block>>>(d_input_array, d_sum_result);
  CUDA_CHECK(hipGetLastError());

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel execution time: %f ms (use NCU for a more precise measurement!)\n", milliseconds);

  // Move data from device to host
  CUDA_CHECK(hipMemcpy(h_sum_result.data(), d_sum_result, sum_memsize, hipMemcpyDeviceToHost));

  // Check values and measure execution time
  printf("Verifying sum...\n");
  if (verify_prefix_sum(h_input_array.data(), h_sum_result.data()) != 0)
  {
    return 1;
  }
  printf("All values match\n");

  // Free memory and destroy events
  CUDA_CHECK(hipFree(d_input_array));
  CUDA_CHECK(hipFree(d_sum_result));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}